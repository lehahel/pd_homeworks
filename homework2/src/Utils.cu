#include "Utils.cuh"
#include <iostream>
namespace utils {

int GetBlockSize() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  return roundl(sqrtl(deviceProp.maxThreadsPerBlock));
}

float* ToDevice(const float* data, size_t size) {
  float* result;
  hipMalloc(&result, size * sizeof(float));
  hipMemcpy(result, data, size * sizeof(float), hipMemcpyHostToDevice);
  return result;
}

PtrWithPitch ToDevice2D(const float* data, size_t width, size_t height) {
  PtrWithPitch result;
  hipMallocPitch(&result.ptr, &result.pitch, width * sizeof(float), height);
  hipMemcpy2D(result.ptr, result.pitch, data, width * sizeof(float),
               width * sizeof(float), height, hipMemcpyHostToDevice);
  return result;
}

float* FromDevice(const float* data, size_t size) {
  float* result = new float[size];
  hipMemcpy(result, data, size * sizeof(float), hipMemcpyDeviceToHost);
  return result;
}

float* FromDevice2D(PtrWithPitch data, size_t width, size_t height) {
  float* result = new float[width * height];
  hipMemcpy2D(result, width * sizeof(float), data.ptr, data.pitch,
               width * sizeof(float), height, hipMemcpyDeviceToHost);
  return result;
}

float* MallocDevice(size_t size) {
  float* result;
  hipMalloc(&result, size * sizeof(float));
  return result;
}

PtrWithPitch MallocDevice2D(size_t width, size_t height) {
  PtrWithPitch result;
  hipMallocPitch(&result.ptr, &result.pitch, width * sizeof(float), height);
  return result;
}

void DeviceFree(float* data) {
  hipFree(data);
}

} // namespace utils
